#include "hip/hip_runtime.h"
#include "Denoise.cuh"


__device__ float generate(hiprandState* globalState, int ind)
{
	hiprandState localState = globalState[ind];
	float RANDOM = hiprand_uniform(&localState);
	globalState[ind] = localState;
	return RANDOM;
}

__global__ void setup_kernel(hiprandState* state, unsigned long seed, int width)
{
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int id = y * width + x;
	hiprand_init(seed, id, 0, &state[id]);
}

__global__ void cudaAddNoise(float* image, int width, int height, float* d_cost, hiprandState* globalState)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	float cudaAddNoise;
	if ((i < height) && (j < width))
	{
		cudaAddNoise = image[i * width + j] + generate(globalState, i * width + j) * 20.0f;
		d_cost[i * width + j] = cudaAddNoise * cudaAddNoise;
		image[i * width + j] = (float)cudaAddNoise;
	}
}


hipError_t addGaussianWhiteNoise(float* hostImage, int width, int height)
{
	hipError_t cudaStatus;
	float* cudaImage;
	float* d_cost;
	{
		CudaCall(hipSetDevice(0));

		CudaCall(hipMalloc((void**)&cudaImage, width * height * sizeof(float)));

		CudaCall(hipMalloc((void**)&d_cost, width * height * sizeof(float)));

	} //initCuda and variables

	// Copy input image from host memory to GPU buffers.
	CudaCall(hipMemcpy(cudaImage, hostImage, width * height * sizeof(float), hipMemcpyHostToDevice));


	hiprandState* devStates;
	hipMalloc(&devStates, width * height * sizeof(hiprandState));
	srand(time(0));
	int seed = rand();
	dim3 threadsPerBlock(16, 16);
	dim3 blocksPerGrid((int)ceil(width / 16), (int)ceil(height / 16));
	setup_kernel << <blocksPerGrid, threadsPerBlock >> > (devStates, seed, width);
	cudaAddNoise << <blocksPerGrid, threadsPerBlock >> > (cudaImage, width, height, d_cost, devStates);
	CudaCall(hipMemcpy(hostImage, cudaImage, width * height * sizeof(float), hipMemcpyDeviceToHost));

	// Check for any errors launching the kernel
	CudaCall(hipGetLastError());

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	CudaCall(hipDeviceSynchronize());

	// Copy output vector from GPU buffer to host memory.
	CudaCall(hipMemcpy(hostImage, cudaImage, width * height * sizeof(float), hipMemcpyDeviceToHost));

	CudaCall(hipFree(cudaImage));
	CudaCall(hipFree(d_cost));
}


int AddNoiseToImage(int width, int height, unsigned char* data)
{

	int ImageSize = width * height;
	std::ofstream myfile;

	float* noisyImage = new float[ImageSize];
	for (int i = 0; i < ImageSize; ++i)
	{
		noisyImage[i] = data[i];
	}


	addGaussianWhiteNoise(noisyImage, width, height);

	for (int i = 0; i < ImageSize; ++i)
	{
		data[i] = clamp(noisyImage[i], 0.0f, 255.0f);
	}
	return 0;
}